#include "hip/hip_runtime.h"
﻿
#include <iostream>

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdio>
#include "hipblas.h"

#include "Utils.h"

#define IDX2C(i,j,ld) (((j)*(ld))+(i))
#define iszero(val) (abs(val) < 0.00000001)

#define cudacall(call)                                                                                                          \
    do                                                                                                                          \
    {                                                                                                                           \
        hipError_t err = (call);                                                                                               \
        if(hipSuccess != err)                                                                                                  \
        {                                                                                                                       \
            fprintf(stderr,"CUDA Error:\nFile = %s\nLine = %d\nReason = %s\n", __FILE__, __LINE__, hipGetErrorString(err));    \
            hipDeviceReset();                                                                                                  \
            exit(EXIT_FAILURE);                                                                                                 \
        }                                                                                                                       \
    }                                                                                                                           \
    while (0)

#define cublascall(call)                                                                                        \
    do                                                                                                          \
    {                                                                                                           \
        hipblasStatus_t status = (call);                                                                         \
        if(HIPBLAS_STATUS_SUCCESS != status)                                                                     \
        {                                                                                                       \
            fprintf(stderr,"CUBLAS Error:\nFile = %s\nLine = %d\nCode = %d\n", __FILE__, __LINE__, status);     \
            hipDeviceReset();                                                                                  \
            exit(EXIT_FAILURE);                                                                                 \
        }                                                                                                       \
                                                                                                                \
    }                                                                                                           \
    while(0)


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

	//const int arraySize = 5;
    //const int a[arraySize] = { 1, 2, 3, 4, 5 };
    //const int b[arraySize] = { 10, 20, 30, 40, 50 };
    //int c[arraySize] = { 0 };
    //
    //// Add vectors in parallel.
    //hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addWithCuda failed!");
    //    return 1;
    //}
    //
    //printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
    //    c[0], c[1], c[2], c[3], c[4]);
    //
    //// hipDeviceReset must be called before exiting in order for profiling and
    //// tracing tools such as Nsight and Visual Profiler to show complete traces.
    //cudaStatus = hipDeviceReset();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipDeviceReset failed!");
    //    return 1;
    //}

int main()
{
    hipblasHandle_t handle;
    cublascall(hipblasCreate(&handle));

    int n_rows, n_cols, nnz;
    int *csc_col_ptr_A, *csc_row_ind_A, *csc_col_ptr_M, *csc_row_ind_M;
    double* csc_val_A, *csc_val_M;

    try
    {
        const char* file_name = "../sherman1.mtx";
        Utils::read_matrix_market_file_col_major_sparse(file_name, n_rows, n_cols, nnz, csc_col_ptr_A,
            csc_val_A, csc_row_ind_A);
        printf("%s, %d x %d matrix loaded.\n", file_name, n_rows, n_cols);
    }
    catch (const std::exception&)
    {
        return EXIT_FAILURE;
    }

    Utils::create_identity_csc(n_rows, csc_col_ptr_M, csc_val_M, csc_row_ind_M);

    for (int k = 0; k < n_cols; k++)
    {
	    const int beg = csc_col_ptr_M[k]; // points to start of sub-array with column k
	    const int end = csc_col_ptr_M[k + 1]; // points to end of sub-array with column k

        // Construct J by getting all row indices from beg to end 
        int n2 = end - beg;
        int* J = static_cast<int*>(malloc(sizeof(int) * n2));
        for (int j = 0; j < n2; j++)
        {
            J[j] = csc_row_ind_M[j + beg];
        }

        // Construct I: nonzero rows of A[:,J]
        // We have to allocate n2 = reduce (+) (map (\j -> shape_A[j]) J) = reduce (+) (map (\j -> csc_col_ptr_A[j+1]-csc_col_ptr_A[j]) J)
        int n1 = 0;
    	for (int j = 0; j < n2; j++)
        {
            const int col_ind = J[j];
            n1 += csc_col_ptr_A[col_ind + 1] - csc_col_ptr_A[col_ind];
        }

        // Get indices from csc_row_ind_A starting from the column pointers from csc_col_ptr_A[J]
        // Construct dense A[I,J] in column major format to be used in batched QR decomposition.
        int* I = static_cast<int*>(malloc(sizeof(int) * n1));
        double* A_hat = static_cast<double*>(malloc(sizeof(double) * n1 * n2));
        int i_ind = 0;
        for (int j = 0; j < n2; j++)
        {
            const int col_ind = J[j];
            const int col_start = csc_col_ptr_A[col_ind];
            const int col_end = csc_col_ptr_A[col_ind + 1];
            for (int i = col_start; i < col_end; i++)
            {
                I[i_ind] = csc_row_ind_A[i];
                A_hat[IDX2C(i_ind, j, n1)] = csc_val_A[i];
                i_ind++;
            }
        }

        // Compute QR decomposition of A_hat
        double* devA_hat;
        cudacall(hipMalloc((void**)&devA_hat, n1 * n2 * sizeof(*devA_hat)));
        cudacall(hipMemcpy(devA_hat, A_hat, n1 * n2 * sizeof(devA_hat), hipMemcpyHostToDevice));

        // Free all allocations
        free(A_hat);
        free(I);
        free(J);
        cudacall(hipFree(devA_hat));
    }


    free(csc_col_ptr_A); free(csc_val_A); free(csc_row_ind_A);
    free(csc_col_ptr_M); free(csc_val_M); free(csc_row_ind_M);
    hipblasDestroy(handle);
    return 0;
}






































// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
