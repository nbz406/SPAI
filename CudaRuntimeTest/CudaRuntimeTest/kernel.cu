#include "hip/hip_runtime.h"
﻿
#include <iostream>

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdio>
#include "hipblas.h"

#include "Utils.h"

#define IDX2C(i,j,ld) (((j)*(ld))+(i))
#define iszero(val) (abs(val) < 0.00000001)

template <typename T> int sgn(T val) {
    return (T(0) < val) - (val < T(0));
}

#define cudacall(call)                                                                                                          \
    do                                                                                                                          \
    {                                                                                                                           \
        hipError_t err = (call);                                                                                               \
        if(hipSuccess != err)                                                                                                  \
        {                                                                                                                       \
            fprintf(stderr,"CUDA Error:\nFile = %s\nLine = %d\nReason = %s\n", __FILE__, __LINE__, hipGetErrorString(err));    \
            hipDeviceReset();                                                                                                  \
            exit(EXIT_FAILURE);                                                                                                 \
        }                                                                                                                       \
    }                                                                                                                           \
    while (0)

#define cublascall(call)                                                                                        \
    do                                                                                                          \
    {                                                                                                           \
        hipblasStatus_t status = (call);                                                                         \
        if(HIPBLAS_STATUS_SUCCESS != status)                                                                     \
        {                                                                                                       \
            fprintf(stderr,"CUBLAS Error:\nFile = %s\nLine = %d\nCode = %d\n", __FILE__, __LINE__, status);     \
            hipDeviceReset();                                                                                  \
            exit(EXIT_FAILURE);                                                                                 \
        }                                                                                                       \
                                                                                                                \
    }                                                                                                           \
    while(0)


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

	//const int arraySize = 5;
    //const int a[arraySize] = { 1, 2, 3, 4, 5 };
    //const int b[arraySize] = { 10, 20, 30, 40, 50 };
    //int c[arraySize] = { 0 };
    //
    //// Add vectors in parallel.
    //hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addWithCuda failed!");
    //    return 1;
    //}
    //
    //printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
    //    c[0], c[1], c[2], c[3], c[4]);
    //
    //// hipDeviceReset must be called before exiting in order for profiling and
    //// tracing tools such as Nsight and Visual Profiler to show complete traces.
    //cudaStatus = hipDeviceReset();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipDeviceReset failed!");
    //    return 1;
    //}

int main()
{
    hipblasHandle_t handle;
    cublascall(hipblasCreate(&handle));

    int n_rows, n_cols, nnz;
    int *csc_col_ptr_A, *csc_row_ind_A, *csc_col_ptr_M, *csc_row_ind_M;
    double* csc_val_A, *csc_val_M;

    try
    {
        const char* file_name = "../orsirr_2.mtx";
        Utils::read_matrix_market_file_col_major_sparse(file_name, n_rows, n_cols, nnz, csc_col_ptr_A,
            csc_val_A, csc_row_ind_A);
        printf("%s, %d x %d matrix loaded.\n", file_name, n_rows, n_cols);
    }
    catch (const std::exception&)
    {
        return EXIT_FAILURE;
    }

    Utils::create_identity_csc(n_rows, csc_col_ptr_M, csc_val_M, csc_row_ind_M);

    int Js_size = 0;
    int Is_size = 0;
    int A_hats_size = 0;

    int* n2s = static_cast<int*>(malloc(sizeof(int) * n_cols));
    int* n1s = static_cast<int*>(malloc(sizeof(int) * n_cols));
    int* Js_inds = static_cast<int*>(malloc(sizeof(int) * n_cols));
    int* Is_inds = static_cast<int*>(malloc(sizeof(int) * n_cols));
    int* A_hats_inds = static_cast<int*>(malloc(sizeof(int) * n_cols));

    for (int k = 0; k < n_cols; k++)
    {
        const int beg = csc_col_ptr_M[k]; // points to start of sub-array with column k
        const int end = csc_col_ptr_M[k + 1]; // points to end of sub-array with column k

        // Construct J by getting all row indices from beg to end
        const int n2 = end - beg;
        n2s[k] = n2;
    }

    // scan n2s
    for (int k = 0; k < n_cols; k++)
    {
        Js_inds[k] = Js_size;
        Js_size += n2s[k];
    }

    int* Js = static_cast<int*>(malloc(sizeof(int) * Js_size));

    for (int k = 0; k < n_cols; k++)
    {
        const int beg = csc_col_ptr_M[k]; // points to start of sub-array with column k
        for (int j = 0; j < n2s[k]; j++)
        {
            Js[Js_inds[k]] = csc_row_ind_M[j + beg];
        }

        // Construct I: nonzero rows of A[:,J]
        // We have to allocate n2 = reduce (+) (map (\j -> shape_A[j]) J) = reduce (+) (map (\j -> csc_col_ptr_A[j+1]-csc_col_ptr_A[j]) J)
        int n1 = 0;
        for (int j = 0; j < n2s[k]; j++)
        {
            const int col_ind = Js[Js_inds[k]];
            n1 += csc_col_ptr_A[col_ind + 1] - csc_col_ptr_A[col_ind];
        }
        n1s[k] = n1;
    }

    int Qs_size = 0;
    // Scan n1s
    int* Qs_inds = static_cast<int*>(malloc(sizeof(int) * n_cols));

    for (int k = 0; k < n_cols; k++)
    {
        const int n1 = n1s[k];
        Is_inds[k] = Is_size;
        Is_size += n1;
        A_hats_inds[k] = A_hats_size;
        A_hats_size += n1 * n2s[k];
        Qs_inds[k] = Qs_size;
        Qs_size += n1 * n1;
    }


    int* ind_of_col_k_in_I_k = static_cast<int*>(malloc(sizeof(int)*n_cols));
    int* Is = static_cast<int*>(malloc(sizeof(int) * Is_size));
    double* A_hats = static_cast<double*>(malloc(sizeof(double) * A_hats_size));

    for (int k = 0; k < n_cols; k++)
    {
        // Get indices from csc_row_ind_A starting from the column pointers from csc_col_ptr_A[J]
        // Construct dense A[I,J] in column major format to be used in batched QR decomposition.
        for (int j = 0; j < n2s[k]; j++)
        {
            const int col_ind = Js[Js_inds[k]];
            const int col_start = csc_col_ptr_A[col_ind];
            const int col_end = csc_col_ptr_A[col_ind + 1];
            for (int i = col_start; i < col_end; i++)
            {
                const int row_ind = csc_row_ind_A[i];
                const int I_ind = i - col_start;
                if (row_ind == k)
                {
                    // Keep track of index of column k in I_k
                    ind_of_col_k_in_I_k[k] = I_ind;
                }
                Is[Is_inds[k] + I_ind] = row_ind;
                A_hats[A_hats_inds[k] + IDX2C(i - col_start, j, n1s[k])] = csc_val_A[i];
            }
        }
    }

    // Householder QR decomposition in place, will only need space for diagonal of Ahat: min(n1,n2)
    int* alphas_inds = static_cast<int*>(malloc(sizeof(int) * n_cols));
    int alphas_size = 0;
    for (int k = 0; k < n_cols; k++)
    {
        alphas_inds[k] = alphas_size;
        alphas_size += std::min(n1s[k], n2s[k]);
    }

    double* alphas = static_cast<double*>(malloc(sizeof(double) * alphas_size));

    for (int k = 0; k < n_cols; k++) {
        const int A_hat_ind = A_hats_inds[k];
        const int n1 = n1s[k];
        const int n2 = n2s[k];
        const int p = std::min(n1, n2);
        for (int j = 0; j < p; j++)
        {
            //alpha[j]=np.linalg.norm(A[j:,j])*np.sign(A[j,j])
            double A_col_norm_squared = 0;
            const int sign = sgn(A_hats[A_hat_ind]);
            for (int i = j; i < n1; i++)
            {
                A_col_norm_squared += pow(A_hats[A_hat_ind + IDX2C(i, j, n1)], 2);
            }
            const double alpha_j = sign * sqrt(A_col_norm_squared);
            alphas[alphas_inds[k] + j] = alpha_j;

            if (!iszero(alpha_j))
            {
                const double A_jj = A_hats[A_hat_ind + IDX2C(j, j, n1)];
                const double beta = 1 / sqrt(2 * alpha_j * (alpha_j + A_jj));
                A_hats[A_hat_ind + IDX2C(j, j, n1)] = beta * (A_jj + alpha_j);
                //A[j+1:,j]=beta*A[j+1:,j] : rest of column multiply by beta.
                for (int i = j + 1; i < n1; i++)
                {
                    A_hats[A_hat_ind + IDX2C(i, j, n1)] *= beta;
                }
                for (int l = j + 1; l < n1; l++)
                {
                    // vTA = A[j:,l].T * A[j:,j]
                    double vTA = 0;
                    for (int i = j; i < n1; i++)
                    {
                        vTA += A_hats[A_hat_ind + IDX2C(i, l, n1)] * A_hats[A_hat_ind + IDX2C(i, j, n1)];
                    }
                    for (int i = j; i < n1; i++)
                    {
                        A_hats[A_hat_ind + IDX2C(i, l, n1)] = A_hats[A_hat_ind + IDX2C(i, l, n1)] - 2 * A_hats[A_hat_ind + IDX2C(i, j, n1)] * vTA;
                    }
                }
            }
        }
    }

    // construct Q (needed for update) and R in place (in A_hats


	
    for (int k = 0; k < n_cols; k++)
    {
        const int batchsize = 1;
        const int ltau = std::max(1, std::min(n1s[k], n2s[k]));

        double* dA_hat;     cudacall(hipMalloc((void**)&dA_hat, batchsize * n1s[k] * n2s[k] * sizeof(*dA_hat)));
        cudacall(hipMemcpy(dA_hat, &A_hats[A_hats_inds[k]], n1s[k] * n2s[k] * sizeof(dA_hat), hipMemcpyHostToDevice));

        double* d_TAU;      cudacall(hipMalloc((void**)&d_TAU, batchsize * ltau * sizeof(double)));

        double* h_A_hatArray[batchsize], * h_TauArray[batchsize];

        for (int i = 0; i < batchsize; i++)
        {
            h_A_hatArray[i] = dA_hat + i * n1s[k] * n2s[k];
            h_TauArray[i] = d_TAU + i * ltau;
        }

        double** d_Aarray, ** d_TauArray;
        hipMalloc((void**)&d_Aarray, sizeof(h_A_hatArray));
        hipMalloc((void**)&d_TauArray, sizeof(h_TauArray));

        hipMemcpy(d_Aarray, h_A_hatArray, sizeof(h_A_hatArray), hipMemcpyHostToDevice);
        hipMemcpy(d_TauArray, h_TauArray, sizeof(h_TauArray), hipMemcpyHostToDevice);
        int info;
        cublascall(hipblasDgeqrfBatched(handle, n1s[k], n2s[k], d_Aarray, n1s[k], d_TauArray, &info, batchsize));

        double* tau = static_cast<double*>(malloc(sizeof(double) * ltau));
        cudacall(hipMemcpy(&A_hats[A_hats_inds[k]], dA_hat, batchsize * n1s[k] * n2s[k] * sizeof(double), hipMemcpyDeviceToHost));
        cudacall(hipMemcpy(tau, d_TAU, batchsize * ltau * sizeof(double), hipMemcpyDeviceToHost));



        // Free all allocations
        cudacall(hipFree(dA_hat));
        cudacall(hipFree(d_TAU));
        cudacall(hipFree(d_Aarray));
        cudacall(hipFree(d_TauArray));
    }

    free(A_hats_inds); free(A_hats);
    free(n1s); free(Js); free(Js_inds);
    free(n2s); free(Is); free(Is_inds);
    free(csc_col_ptr_A); free(csc_val_A); free(csc_row_ind_A);
    free(csc_col_ptr_M); free(csc_val_M); free(csc_row_ind_M);
    hipblasDestroy(handle);
    return 0;
}






































// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
