#include "hip/hip_runtime.h"
﻿
#include <iostream>

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdio>
#include <vector>
#include <algorithm>
#include <chrono>
#include <hipblas.h>
#include <hipsparse.h>
#include <hipblaslt.h>

#include "Utils.h"

#define IDX2C(i,j,ld) (((j)*(ld))+(i))
#define iszero(val) (abs(val) < 0.00000001)

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

void sequential_spai(double* A, double* M, const int N)
{
    double* e_k = static_cast<double*>(malloc(N * N * sizeof(double)));

    // Populate M with ones in diagonal. Column major, so outer loop is over columns. Also populates e_k.
    for (int j = 0; j < N; j++)
    {
        for (int i = 0; i < N; i++)
        {
            if (i == j) {
                M[IDX2C(i, j, N)] = 1;
				e_k[IDX2C(i, j, N)] = 1;
			}
            else
            {
                M[IDX2C(i, j, N)] = 0;
                e_k[IDX2C(i, j, N)] = 0;
            }
        }
    }
    int nnz = 0;
    // for each column
    for (int k = 0; k < N; k++)
    {
        int iteration = 0;
        // each column is M from IDX2C(0, k, N) to IDX2C(0, k, N-1);
        // e_k[IDX2C(1 to N-1, k, N)]
        
        // Calculate J, an array of nonzero row indices of M
        std::vector<int> J;
        for (int i = 0; i < N; i++)
        {
            if (!iszero(M[IDX2C(i,k,N)]))
                J.emplace_back(i);
        }
        // Column major CSR can get indices from offsets array.

    }
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }


    int n_rows, n_cols, nnz;
    int *csc_col_ptr_A, *csc_row_ind_A, *csc_col_ptr_M, *csc_row_ind_M;
    double* csc_val_A, *csc_val_M;

    try
    {
        const char* file_name = "../sherman1.mtx";
        Utils::read_matrix_market_file_col_major_sparse(file_name, n_rows, n_cols, nnz, csc_col_ptr_A,
            csc_val_A, csc_row_ind_A);
        printf("%s, %d x %d matrix loaded.\n", file_name, n_rows, n_cols);
    }
    catch (const std::exception&)
    {
        return EXIT_FAILURE;
    }

    Utils::create_identity_csc(n_rows, csc_col_ptr_M, csc_val_M, csc_row_ind_M);

    for (int k = 0; k < n_cols; k++)
    {
	    const int beg = csc_col_ptr_M[k]; // points to start of sub-array with column k
	    const int end = csc_col_ptr_M[k + 1]; // points to end of sub-array with column k

        // Construct J by getting all row indices from beg to end 
        int n2 = end - beg;
        int* J = static_cast<int*>(malloc(sizeof(int) * n2));
        for (int j = 0; j < n2; j++)
        {
            J[j] = csc_row_ind_M[j + beg];
        }

        // Construct I: nonzero rows of A[:,J]
        // We have to allocate n2 = reduce (+) (map (\j -> shape_A[j]) J) = reduce (+) (map (\j -> csc_col_ptr_A[j+1]-csc_col_ptr_A[j]) J)
        int n1 = 0;
    	for (int j = 0; j < n2; j++)
        {
            const int col_ind = J[j];
            n1 += csc_col_ptr_A[col_ind + 1] - csc_col_ptr_A[col_ind];
        }

        // get indices from csc_row_ind_A starting from the column pointers from csc_col_ptr_A[J]
        int* I = static_cast<int*>(malloc(sizeof(int) * n1));
        int i_ind = 0;
        for (int j = 0; j < n2; j++)
        {
            const int col_ind = J[j];
            for (int i = csc_col_ptr_A[col_ind]; i < csc_col_ptr_A[col_ind + 1]; i++)
            {
                I[i_ind] = csc_row_ind_A[i];
                i_ind++;
            }
        }

        // Construct dense A[I,J] in column major format to be used in batched QR decomposition.
        double* A_hat = static_cast<double*>(malloc(sizeof(double) * n1 * n2));
        for (int j = 0; j < n2; j++)
        {
            const int col_ind = J[j];
            const int col_start = csc_col_ptr_A[col_ind];
            const int col_end = csc_col_ptr_A[col_ind + 1];
            int* beg = &csc_row_ind_A[col_start];
            int* end = &csc_row_ind_A[col_end];
	        for (int i = 0; i < n1; i++)
	        {
                const int row_ind = I[i];
                // Need to search for row ind I[i] in csc_row_ind_A: idx and return the val at this idx.
                // Indices in csc_row_ind_A are sorted per column, so binary search works
                int* itr = std::lower_bound(beg, end, row_ind);
                const int index = std::distance(beg, itr);
                A_hat[IDX2C(i, j, n1)] = csc_val_A[col_start + index];
	        }
        }
        for (int i = 0; i < n1; i++)
        {
            for (int j = 0; j < n2; j++)
	        {
                printf(" %e ", A_hat[IDX2C(i, j, n1)]);
	        }
            printf("\n");
        }
        printf("\n");

        // Free all allocations
        free(A_hat);
        free(I);
        free(J);
    }


    free(csc_col_ptr_A); free(csc_val_A); free(csc_row_ind_A);
    free(csc_col_ptr_M); free(csc_val_M); free(csc_row_ind_M);
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
